#include "src/Filters.cuh"
#include <chrono>

int main() {
	FILE *outputFile = NULL;
	if (OUTPUT_FILEPATH) {
		outputFile = std::fopen(OUTPUT_FILEPATH, "w");
		if (!outputFile) ABORT("ERROR: Failed to open %s.\n", OUTPUT_FILEPATH);
	} else if (SILENT_MODE) ABORT("ERROR: No output method for results was provided (SILENT_MODE is enabled and no filepath was specified).\n");

	if (!SILENT_MODE) {
		POPULATION_CHUNKS_DATA.printFindings();
		constexpr double inputDataBits = POPULATION_CHUNKS_DATA.getEstimatedBits();
		if (inputDataBits < 48.) std::fprintf(stderr, "WARNING: The input data very likely does not have enough information to reduce the search space to a single structure seed (%.2g/48 bits).\nIt is VERY HIGHLY recommended you gather more data and only afterwards run the program.\n\n", inputDataBits);
		else {
			constexpr double inputDataHighestPopulationChunkBits = POPULATION_CHUNKS_DATA.treeChunks[0].getEstimatedBits();
			if (inputDataHighestPopulationChunkBits < 48.) std::fprintf(stderr, "\n\nWARNING: The input data's highest-information population chunk very likely does not have enough information to reduce the search space to a single structure seed by itself (%.2g/48 bits).\nOther chunks will be used later to account for this, but this program will run faster and use less memory if you gather more data and only afterwards run the program.\n", inputDataHighestPopulationChunkBits);
		}
	}

	void *theoreticalAttributesFilterMasksPointer;
	TRY_CUDA(hipGetSymbolAddress(&theoreticalAttributesFilterMasksPointer, HIP_SYMBOL(filter3_masks)));

	#if (!CUDA_IS_PRESENT)
		threads = static_cast<pthread_t *>(malloc(NUMBER_OF_WORKERS*sizeof(*threads)));
		__numberOfThreads = NUMBER_OF_WORKERS;
		ThreadData data[NUMBER_OF_WORKERS];
		for (uint64_t i = 0; i < NUMBER_OF_WORKERS; ++i) data[i].index = i;
	#endif

	auto startTime = std::chrono::steady_clock::now(), currentTime = startTime;
	for (uint64_t partialRun = ACTUAL_PARTIAL_RUN_TO_BEGIN_FROM; partialRun <= ACTUAL_NUMBER_OF_PARTIAL_RUNS; ++partialRun) {
		// TODO: Rework to avoid overflows
		uint64_t runStartSeed = constexprRound(static_cast<double>((partialRun - 1)*TOTAL_NUMBER_OF_STATES_TO_CHECK)/static_cast<double>(ACTUAL_NUMBER_OF_PARTIAL_RUNS));
		uint64_t runEndSeed = constexprRound(static_cast<double>(partialRun*TOTAL_NUMBER_OF_STATES_TO_CHECK)/static_cast<double>(ACTUAL_NUMBER_OF_PARTIAL_RUNS));
		if (!SILENT_MODE) std::fprintf(stderr, "Beginning partial run #%" PRIu64 " of %" PRIu64 "\t(states [%" PRIu64 ", %" PRIu64 "] out of %" PRIu64 ").\n", partialRun, ACTUAL_NUMBER_OF_PARTIAL_RUNS, runStartSeed, runEndSeed - 1, TOTAL_NUMBER_OF_STATES_TO_CHECK - 1);
		for (uint64_t runCurrentSeed = runStartSeed; runCurrentSeed < runEndSeed; runCurrentSeed += NUMBER_OF_WORKERS) {
			if (!SILENT_MODE && TIME_PROGRAM && !(runCurrentSeed/NUMBER_OF_WORKERS & 255)) {
				auto lastTime = currentTime;
				currentTime = std::chrono::steady_clock::now();
				double totalSeconds = std::chrono::duration_cast<std::chrono::nanoseconds>(currentTime - startTime).count()/1e9;
				if (totalSeconds > 0.2) { // So we don't get a timestamp with meaningless values for the very first, 0-second entry
					double durationSeconds = std::chrono::duration_cast<std::chrono::nanoseconds>(currentTime - lastTime).count()/1e9;
					double seedsPerSecond = NUMBER_OF_WORKERS/durationSeconds;
					double eta = (TOTAL_NUMBER_OF_STATES_TO_CHECK - (runCurrentSeed + NUMBER_OF_WORKERS))/seedsPerSecond;
					std::fprintf(stderr, "%.3f seconds\t%.3f billion seeds/second\tETA: %f seconds\n", totalSeconds, seedsPerSecond/1e9, eta);
				}
			}
			
			if (filter2_numberOfResultsPerRun) {
				std::fprintf(stderr, "Filter 1: %" PRIu64 " results\n", filter1_numberOfResultsPerRun);
				std::fprintf(stderr, "Filter 2: %" PRIu64 " results\n", filter2_numberOfResultsPerRun);
				std::fprintf(stderr, "Filter 3: %" PRIu64 " results\n", filter3_numberOfResultsPerRun);
				std::fprintf(stderr, "Filter 4: %" PRIu64 " results\n", treechunkFilter_numberOfResultsPerRun);
				std::fprintf(stderr, "Filter 5: %" PRIu64 " results\n", totalStructureSeedsPerRun);
				std::fprintf(stderr, "Filter 6: %" PRIu64 " results\n", totalWorldseedsPerRun);
			}
			
			filter1_numberOfResultsPerRun = filter2_numberOfResultsPerRun = filter3_numberOfResultsPerRun = treechunkFilter_numberOfResultsPerRun = totalStructureSeedsPerRun = totalWorldseedsPerRun = 0;
			currentPopulationChunkDataIndex = 0;

			// NelS: Can this be moved?
			if (COLLAPSE_NEARBY_SEEDS_FLAG) hipMemsetAsync(theoreticalAttributesFilterMasksPointer, 0, sizeof(filter3_masks));

			#if CUDA_IS_PRESENT
				// filter1<<<constexprCeil(static_cast<double>(NUMBER_OF_WORKERS)/static_cast<double>(ACTUAL_WORKERS_PER_BLOCK)), ACTUAL_WORKERS_PER_BLOCK>>>(runCurrentSeed);
				filter1<<<NUMBER_OF_WORKERS/ACTUAL_WORKERS_PER_BLOCK + 1, ACTUAL_WORKERS_PER_BLOCK>>>(runCurrentSeed);
			#else
				for (uint64_t i = 0; i < NUMBER_OF_WORKERS; ++i) {
					data[i].start = runCurrentSeed;
					pthread_create(&threads[i], NULL, filter1, &data[i]);
				}
			#endif
			TRY_CUDA(hipGetLastError());
			TRY_CUDA(hipDeviceSynchronize());

			if (filter1_numberOfResultsPerRun > ACTUAL_MAX_NUMBER_OF_RESULTS_PER_RUN) {
				if (!SILENT_MODE) std::fprintf(stderr, "WARNING: Filter 1 caught more results (%" PRIu64 ") than maximum number allowed (%" PRIu64 "); ignoring last %" PRIu64 " results.\n", filter1_numberOfResultsPerRun, ACTUAL_MAX_NUMBER_OF_RESULTS_PER_RUN, filter1_numberOfResultsPerRun - ACTUAL_MAX_NUMBER_OF_RESULTS_PER_RUN);
				filter1_numberOfResultsPerRun = ACTUAL_MAX_NUMBER_OF_RESULTS_PER_RUN;
			}
			if (!filter1_numberOfResultsPerRun) continue;

			// for (uint64_t i = 0; i < filter1_numberOfResultsPerRun; ++i) {

			// }

			TRY_CUDA(hipMemcpy(filterInputs, filterResults, sizeof(*filterResults)*filter1_numberOfResultsPerRun, hipMemcpyDefault));
			#if CUDA_IS_PRESENT
				// filter2<<<constexprCeil(static_cast<double>(filter1_numberOfResultsPerRun)/static_cast<double>(ACTUAL_WORKERS_PER_BLOCK)), ACTUAL_WORKERS_PER_BLOCK>>>();
				filter2<<<filter1_numberOfResultsPerRun/ACTUAL_WORKERS_PER_BLOCK + 1, ACTUAL_WORKERS_PER_BLOCK>>>();
			#else
				for (uint64_t i = 0; i < filter1_numberOfResultsPerRun; ++i) pthread_create(&threads[i], NULL, filter2, &data[i]);
			#endif
			TRY_CUDA(hipGetLastError());
			TRY_CUDA(hipDeviceSynchronize());

			if (filter2_numberOfResultsPerRun > ACTUAL_MAX_NUMBER_OF_RESULTS_PER_RUN) {
				if (!SILENT_MODE) std::fprintf(stderr, "WARNING: Theoretical-Coords-and-Types filter caught more results (%" PRIu64 ") than maximum number allowed (%" PRIu64 "); ignoring last %" PRIu64 " results.\n", filter2_numberOfResultsPerRun, ACTUAL_MAX_NUMBER_OF_RESULTS_PER_RUN, filter2_numberOfResultsPerRun - ACTUAL_MAX_NUMBER_OF_RESULTS_PER_RUN);
				filter2_numberOfResultsPerRun = ACTUAL_MAX_NUMBER_OF_RESULTS_PER_RUN;
			}
			if (!filter2_numberOfResultsPerRun) continue;

			TRY_CUDA(hipMemcpy(filterInputs, filterResults, sizeof(*filterResults)*filter2_numberOfResultsPerRun, hipMemcpyDefault));
			#if CUDA_IS_PRESENT
				// filter3<<<constexprCeil(static_cast<double>(filter2_numberOfResultsPerRun)/static_cast<double>(ACTUAL_WORKERS_PER_BLOCK)), ACTUAL_WORKERS_PER_BLOCK>>>();
				filter3<<<filter2_numberOfResultsPerRun/ACTUAL_WORKERS_PER_BLOCK + 1, ACTUAL_WORKERS_PER_BLOCK>>>();
			#else
				for (uint64_t i = 0; i < filter2_numberOfResultsPerRun; ++i) pthread_create(&threads[i], NULL, filter3, &data[i]);
			#endif
			TRY_CUDA(hipGetLastError());
			TRY_CUDA(hipDeviceSynchronize());

			if (filter3_numberOfResultsPerRun > ACTUAL_MAX_NUMBER_OF_RESULTS_PER_RUN) {
				if (!SILENT_MODE) std::fprintf(stderr, "WARNING: Theoretical-Attributes filter caught more results (%" PRIu64 ") than maximum number allowed (%" PRIu64 "); ignoring last %" PRIu64 " results.\n", filter3_numberOfResultsPerRun, ACTUAL_MAX_NUMBER_OF_RESULTS_PER_RUN, filter3_numberOfResultsPerRun - ACTUAL_MAX_NUMBER_OF_RESULTS_PER_RUN);
				filter3_numberOfResultsPerRun = ACTUAL_MAX_NUMBER_OF_RESULTS_PER_RUN;
			}
			if (!filter3_numberOfResultsPerRun) continue;

			TRY_CUDA(hipMemcpy(filterInputs, filterResults, sizeof(*filterResults)*filter3_numberOfResultsPerRun, hipMemcpyDefault));
			#if CUDA_IS_PRESENT
				// treechunkFilter<<<constexprCeil(static_cast<double>(filter3_numberOfResultsPerRun * (MAX_CALLS + 1) * (UINT64_C(1) << POPULATION_CHUNKS_DATA.treeChunks[currentPopulationChunkDataIndex].maxTreeCount))/static_cast<double>(ACTUAL_WORKERS_PER_BLOCK)), ACTUAL_WORKERS_PER_BLOCK>>>();
				treechunkFilter<<<filter3_numberOfResultsPerRun * (MAX_CALLS + 1) * (UINT64_C(1) << POPULATION_CHUNKS_DATA.treeChunks[currentPopulationChunkDataIndex].maxTreeCount)/ACTUAL_WORKERS_PER_BLOCK + 1, ACTUAL_WORKERS_PER_BLOCK>>>();
			#else
				for (uint64_t i = 0; i < filter3_numberOfResultsPerRun; ++i) pthread_create(&threads[i], NULL, treechunkFilter, &data[i]); // How to handle larger number of i's?
			#endif
			TRY_CUDA(hipGetLastError());
			TRY_CUDA(hipDeviceSynchronize());

			if (treechunkFilter_numberOfResultsPerRun > ACTUAL_MAX_NUMBER_OF_RESULTS_PER_RUN) {
				if (!SILENT_MODE) std::fprintf(stderr, "WARNING: All-Attributes filter caught more results (%" PRIu64 ") than maximum number allowed (%" PRIu64 "); ignoring last %" PRIu64 " results.\n", treechunkFilter_numberOfResultsPerRun, ACTUAL_MAX_NUMBER_OF_RESULTS_PER_RUN, treechunkFilter_numberOfResultsPerRun - ACTUAL_MAX_NUMBER_OF_RESULTS_PER_RUN);
				treechunkFilter_numberOfResultsPerRun = ACTUAL_MAX_NUMBER_OF_RESULTS_PER_RUN;
			}
			if (!treechunkFilter_numberOfResultsPerRun) continue;

			TRY_CUDA(hipMemcpy(filterInputs, filterResults, sizeof(*filterResults)*treechunkFilter_numberOfResultsPerRun, hipMemcpyDefault));
			// 1.6.4:         ?
			// 1.8.9:         ?
			// 1.12.2:        ./main input.txt output.txt 1 1.12 treechunkXChunkCoordinate treechunkXChunkCoordinate treechunkZChunkCoordinate treechunkZChunkCoordinate 0 10000
			// 1.14.4:        ./main input.txt output.txt 1 1.13 60001 treechunkXChunkCoordinate treechunkXChunkCoordinate treechunkZChunkCoordinate treechunkZChunkCoordinate 0 0
			// 1.16.1/1.16.4: ./main input.txt output.txt 1 1.13 80001 treechunkXChunkCoordinate treechunkXChunkCoordinate treechunkZChunkCoordinate treechunkZChunkCoordinate 0 0
			#if CUDA_IS_PRESENT
				// reversePopulationSeeds<<<constexprCeil(static_cast<double>(treechunkFilter_numberOfResultsPerRun)/static_cast<double>(ACTUAL_WORKERS_PER_BLOCK)), ACTUAL_WORKERS_PER_BLOCK>>>();
				reversePopulationSeeds<<<treechunkFilter_numberOfResultsPerRun/ACTUAL_WORKERS_PER_BLOCK + 1, ACTUAL_WORKERS_PER_BLOCK>>>();
			#else
				for (uint64_t i = 0; i < treechunkFilter_numberOfResultsPerRun; ++i) pthread_create(&threads[i], NULL, reversePopulationSeeds, &data[i]);
			#endif
			TRY_CUDA(hipGetLastError());
			TRY_CUDA(hipDeviceSynchronize());

			if (totalStructureSeedsPerRun > ACTUAL_MAX_NUMBER_OF_RESULTS_PER_RUN) {
				if (!SILENT_MODE) std::fprintf(stderr, "WARNING: Population seed reverser caught more results (%" PRIu64 ") than maximum number allowed (%" PRIu64 "); ignoring last %" PRIu64 " results.\n", totalStructureSeedsPerRun, ACTUAL_MAX_NUMBER_OF_RESULTS_PER_RUN, totalStructureSeedsPerRun - ACTUAL_MAX_NUMBER_OF_RESULTS_PER_RUN);
				totalStructureSeedsPerRun = ACTUAL_MAX_NUMBER_OF_RESULTS_PER_RUN;
			}
			if (!totalStructureSeedsPerRun) continue;

			for (currentPopulationChunkDataIndex = 1; currentPopulationChunkDataIndex < POPULATION_CHUNKS_DATA.numberOfTreeChunks; ++currentPopulationChunkDataIndex) {
				// TODO: Second.cu filtering with other tree chunks
			}

			// TODO: Change totalStructureSeedsPerRun when second.cu filters are implemented
			TRY_CUDA(hipMemcpy(filterInputs, filterResults, sizeof(*filterResults)*totalStructureSeedsPerRun, hipMemcpyDefault));
			for (largeBiomesFlag = 0; largeBiomesFlag <= 1; ++largeBiomesFlag) {
				totalWorldseedsPerRun = 0;
				#if CUDA_IS_PRESENT
					// biomeFilter<<<constexprCeil(static_cast<double>(totalStructureSeedsPerRun*65536)/static_cast<double>(ACTUAL_WORKERS_PER_BLOCK)), ACTUAL_WORKERS_PER_BLOCK>>>();
					biomeFilter<<<totalStructureSeedsPerRun*65536/ACTUAL_WORKERS_PER_BLOCK + 1, ACTUAL_WORKERS_PER_BLOCK>>>();
				#else
					for (uint64_t i = 0; i < totalStructureSeedsPerRun; ++i) pthread_create(&threads[i], NULL, biomeFilter, &data[i]);
				#endif
				TRY_CUDA(hipGetLastError());
				TRY_CUDA(hipDeviceSynchronize());

				if (totalWorldseedsPerRun > ACTUAL_MAX_NUMBER_OF_RESULTS_PER_RUN) {
					if (!SILENT_MODE) std::fprintf(stderr, "WARNING: Biome filter caught more results (%" PRIu64 ") than maximum number allowed (%" PRIu64 "); ignoring last %" PRIu64 " results.\n", totalWorldseedsPerRun, ACTUAL_MAX_NUMBER_OF_RESULTS_PER_RUN, totalWorldseedsPerRun - ACTUAL_MAX_NUMBER_OF_RESULTS_PER_RUN);
					totalWorldseedsPerRun = ACTUAL_MAX_NUMBER_OF_RESULTS_PER_RUN;
				}
				if (totalWorldseedsPerRun) {
					for (uint64_t i = 0; i < totalWorldseedsPerRun; ++i) {
						const char *notes = getNotesAboutSeed(filterResults[i], largeBiomesFlag);
						if (OUTPUT_FILEPATH) std::fprintf(outputFile, "%" PRId64 "%s\n", static_cast<int64_t>(filterResults[i]), notes);
						if (!SILENT_MODE) std::printf("%" PRId64 "%s\n", static_cast<int64_t>(filterResults[i]), notes);
					}
				}
			}
		}
	}

	if (OUTPUT_FILEPATH) std::fclose(outputFile);

	#if (!CUDA_IS_PRESENT)
		free(threads);
	#endif
	return 0;
}