#include "src/Filters.cuh"
#include <chrono>
#include <set>

int main() {
	std::string currentFilepath = OUTPUT_FILEPATH;
	FILE *outputFile = NULL;
	std::string delimiter = "";
	// If an output filepath was specified:
	if (OUTPUT_FILEPATH != NULL && OUTPUT_FILEPATH != "") {
		// First ensure the file doesn't already exist.
		// NOTE: Admittedly vulnerable to TOC/TOU.
		// (Technique adapted from PherricOxide on Stack Overflow (https://stackoverflow.com/a/12774387))
		const std::string stem = getFilepathStem(currentFilepath);
		const std::string extension = getFilepathExtension(currentFilepath);
		for (; (outputFile = std::fopen((stem + delimiter + extension).c_str(), "r")) != NULL; std::fclose(outputFile)) {
			// If it does already exist, add/increment delimiter
			if (delimiter == "") delimiter = "0";
			else delimiter = std::to_string(std::stoi(delimiter) + 1);
		}
		// Set final filepath and warn user if necessary
		currentFilepath = stem + delimiter + extension;

		outputFile = std::fopen(currentFilepath.c_str(), "w");
		if (!outputFile) ABORT("ERROR: Failed to open %s.\n", OUTPUT_FILEPATH);
	// Otherwise if an output filepath wasn't specified:
	} else if (SILENT_MODE) ABORT("ERROR: No output method for results was provided (SILENT_MODE is enabled and no filepath was specified).\n");

	if (!SILENT_MODE) {
		printSettingsAndDataWarnings();
		if (OUTPUT_FILEPATH != currentFilepath) std::fprintf(stderr, "WARNING: The specified output filepath (%s) already exists. The output file has been renamed to %s to avoid overwriting it.\n\n", OUTPUT_FILEPATH, currentFilepath.c_str());
	}

	void *filter3_masksPointer;
	TRY_CUDA(hipGetSymbolAddress(&filter3_masksPointer, HIP_SYMBOL(filter3_masks)));

	#if (!CUDA_IS_PRESENT)
		threads = static_cast<pthread_t *>(malloc(NUMBER_OF_WORKERS*sizeof(*threads)));
		__numberOfThreads = NUMBER_OF_WORKERS;
		ThreadData data[NUMBER_OF_WORKERS];
		for (uint64_t i = 0; i < NUMBER_OF_WORKERS; ++i) data[i].index = i;
	#endif

	auto startTime = std::chrono::steady_clock::now(), currentTime = startTime;
	uint64_t runEndSeed = constexprRound(static_cast<double>(ACTUAL_PARTIAL_RUN_TO_BEGIN_FROM - 1)*static_cast<double>(TOTAL_NUMBER_OF_STATES_TO_CHECK)/static_cast<double>(ACTUAL_NUMBER_OF_PARTIAL_RUNS));
	for (uint64_t partialRun = ACTUAL_PARTIAL_RUN_TO_BEGIN_FROM; partialRun <= ACTUAL_NUMBER_OF_PARTIAL_RUNS; ++partialRun) {
		uint64_t runStartSeed = runEndSeed;
		runEndSeed = constexprRound(static_cast<double>(partialRun)*static_cast<double>(TOTAL_NUMBER_OF_STATES_TO_CHECK)/static_cast<double>(ACTUAL_NUMBER_OF_PARTIAL_RUNS));
		if (!SILENT_MODE) std::fprintf(stderr, "Beginning partial run #%" PRIu64 " of %" PRIu64 "\t(states [%" PRIu64 ", %" PRIu64 "] out of %" PRIu64 ").\n", partialRun, ACTUAL_NUMBER_OF_PARTIAL_RUNS, runStartSeed, runEndSeed - 1, TOTAL_NUMBER_OF_STATES_TO_CHECK - 1);
		for (uint64_t runCurrentSeed = runStartSeed; runCurrentSeed < runEndSeed; runCurrentSeed += NUMBER_OF_WORKERS) {
			if (!SILENT_MODE && PRINT_TIMESTAMPS_FREQUENCY && !(runCurrentSeed/NUMBER_OF_WORKERS % PRINT_TIMESTAMPS_FREQUENCY)) {
				auto lastTime = currentTime;
				currentTime = std::chrono::steady_clock::now();
				double totalSeconds = std::chrono::duration_cast<std::chrono::nanoseconds>(currentTime - startTime).count()/1e9;
				if (totalSeconds > 0.2) { // So we don't get a timestamp with meaningless values for the very first, 0-second entry
					double durationSeconds = std::chrono::duration_cast<std::chrono::nanoseconds>(currentTime - lastTime).count()/1e9;
					double seedsPerSecond = (NUMBER_OF_WORKERS*PRINT_TIMESTAMPS_FREQUENCY)/durationSeconds;
					double eta = (TOTAL_NUMBER_OF_STATES_TO_CHECK - (runCurrentSeed + NUMBER_OF_WORKERS))/seedsPerSecond;
					std::fprintf(stderr, "%.3f seconds\t%.3f billion seeds/second\tETA: %.3f seconds\n", totalSeconds, seedsPerSecond/1e9, eta);
				}
			}
			
			currentPopulationChunkDataIndex = 0;

			if (ABSOLUTE_POPULATION_CHUNKS_DATA.collapseNearbySeedsFlag) hipMemsetAsync(filter3_masksPointer, 0, sizeof(filter3_masks));

			filter1_numberOfResultsPerRun = 0;
			#if CUDA_IS_PRESENT
				// filter1<<<constexprCeil(static_cast<double>(NUMBER_OF_WORKERS)/static_cast<double>(ACTUAL_WORKERS_PER_BLOCK)), ACTUAL_WORKERS_PER_BLOCK>>>(runCurrentSeed);
				filter1<<<NUMBER_OF_WORKERS/ACTUAL_WORKERS_PER_BLOCK + 1, ACTUAL_WORKERS_PER_BLOCK>>>(runCurrentSeed);
			#else
				for (uint64_t i = 0; i < NUMBER_OF_WORKERS; ++i) {
					data[i].start = runCurrentSeed;
					pthread_create(&threads[i], NULL, filter1, &data[i]);
				}
			#endif
			TRY_CUDA(hipGetLastError());
			TRY_CUDA(hipDeviceSynchronize());

			if (filter1_numberOfResultsPerRun > ACTUAL_MAX_NUMBER_OF_RESULTS_PER_RUN) {
				if (!SILENT_MODE) std::fprintf(stderr, "WARNING: Filter 1 caught more results (%" PRIu64 ") than maximum number allowed (%" PRIu64 "); ignoring last %" PRIu64 " results.\n", filter1_numberOfResultsPerRun, ACTUAL_MAX_NUMBER_OF_RESULTS_PER_RUN, filter1_numberOfResultsPerRun - ACTUAL_MAX_NUMBER_OF_RESULTS_PER_RUN);
				filter1_numberOfResultsPerRun = ACTUAL_MAX_NUMBER_OF_RESULTS_PER_RUN;
			}
			if (!filter1_numberOfResultsPerRun) continue;

			
			if (static_cast<void *>(FILTER_2_INPUT) != static_cast<void *>(FILTER_1_OUTPUT)) {
				#if CUDA_IS_PRESENT
					// transferResults<<<constexprCeil(static_cast<double>(filter1_numberOfResultsPerRun)/static_cast<double>(ACTUAL_WORKERS_PER_BLOCK)), ACTUAL_WORKERS_PER_BLOCK>>>(FILTER_1_OUTPUT, FILTER_2_INPUT);
					// transferResults<<<filter1_numberOfResultsPerRun/ACTUAL_WORKERS_PER_BLOCK + 1, ACTUAL_WORKERS_PER_BLOCK>>>(FILTER_1_OUTPUT, FILTER_2_INPUT);
				#else
					for (uint64_t i = 0; i < filter1_numberOfResultsPerRun; ++i) {
						pthread_create(&threads[i], NULL, transferResults, &data[i]);
					}
				#endif
				TRY_CUDA(hipGetLastError());
				TRY_CUDA(hipDeviceSynchronize());
			}

			filter2_numberOfResultsPerRun = 0;
			#if CUDA_IS_PRESENT
				// filter2<<<constexprCeil(static_cast<double>(filter1_numberOfResultsPerRun)/static_cast<double>(ACTUAL_WORKERS_PER_BLOCK)), ACTUAL_WORKERS_PER_BLOCK>>>();
				filter2<<<filter1_numberOfResultsPerRun/ACTUAL_WORKERS_PER_BLOCK + 1, ACTUAL_WORKERS_PER_BLOCK>>>();
			#else
				for (uint64_t i = 0; i < filter1_numberOfResultsPerRun; ++i) pthread_create(&threads[i], NULL, filter2, &data[i]);
			#endif
			TRY_CUDA(hipGetLastError());
			TRY_CUDA(hipDeviceSynchronize());

			if (filter2_numberOfResultsPerRun > ACTUAL_MAX_NUMBER_OF_RESULTS_PER_RUN) {
				if (!SILENT_MODE) std::fprintf(stderr, "WARNING: Theoretical-Coords-and-Types filter caught more results (%" PRIu64 ") than maximum number allowed (%" PRIu64 "); ignoring last %" PRIu64 " results.\n", filter2_numberOfResultsPerRun, ACTUAL_MAX_NUMBER_OF_RESULTS_PER_RUN, filter2_numberOfResultsPerRun - ACTUAL_MAX_NUMBER_OF_RESULTS_PER_RUN);
				filter2_numberOfResultsPerRun = ACTUAL_MAX_NUMBER_OF_RESULTS_PER_RUN;
			}
			if (!filter2_numberOfResultsPerRun) continue;


			if (static_cast<void *>(FILTER_3_INPUT) != static_cast<void *>(FILTER_2_OUTPUT)) {
				#if CUDA_IS_PRESENT
					// transferResults<<<constexprCeil(static_cast<double>(filter2_numberOfResultsPerRun)/static_cast<double>(ACTUAL_WORKERS_PER_BLOCK)), ACTUAL_WORKERS_PER_BLOCK>>>(FILTER_2_OUTPUT, FILTER_3_INPUT);
					// transferResults<<<filter2_numberOfResultsPerRun/ACTUAL_WORKERS_PER_BLOCK + 1, ACTUAL_WORKERS_PER_BLOCK>>>(FILTER_2_OUTPUT, FILTER_3_INPUT);
				#else
					for (uint64_t i = 0; i < filter2_numberOfResultsPerRun; ++i) {
						pthread_create(&threads[i], NULL, transferResults, &data[i]);
					}
				#endif
				TRY_CUDA(hipGetLastError());
				TRY_CUDA(hipDeviceSynchronize());
			}

			filter3_numberOfResultsPerRun = 0;
			#if CUDA_IS_PRESENT
				// filter3<<<constexprCeil(static_cast<double>(filter2_numberOfResultsPerRun)/static_cast<double>(ACTUAL_WORKERS_PER_BLOCK)), ACTUAL_WORKERS_PER_BLOCK>>>();
				filter3<<<filter2_numberOfResultsPerRun/ACTUAL_WORKERS_PER_BLOCK + 1, ACTUAL_WORKERS_PER_BLOCK>>>();
			#else
				for (uint64_t i = 0; i < filter2_numberOfResultsPerRun; ++i) pthread_create(&threads[i], NULL, filter3, &data[i]);
			#endif
			TRY_CUDA(hipGetLastError());
			TRY_CUDA(hipDeviceSynchronize());

			if (filter3_numberOfResultsPerRun > ACTUAL_MAX_NUMBER_OF_RESULTS_PER_RUN) {
				if (!SILENT_MODE) std::fprintf(stderr, "WARNING: Theoretical-Attributes filter caught more results (%" PRIu64 ") than maximum number allowed (%" PRIu64 "); ignoring last %" PRIu64 " results.\n", filter3_numberOfResultsPerRun, ACTUAL_MAX_NUMBER_OF_RESULTS_PER_RUN, filter3_numberOfResultsPerRun - ACTUAL_MAX_NUMBER_OF_RESULTS_PER_RUN);
				filter3_numberOfResultsPerRun = ACTUAL_MAX_NUMBER_OF_RESULTS_PER_RUN;
			}
			if (!filter3_numberOfResultsPerRun) continue;


			if (static_cast<void *>(TREECHUNK_FILTER_INPUT) != static_cast<void *>(FILTER_3_OUTPUT)) {
				#if CUDA_IS_PRESENT
					// transferResults<<<constexprCeil(static_cast<double>(filter3_numberOfResultsPerRun)/static_cast<double>(ACTUAL_WORKERS_PER_BLOCK)), ACTUAL_WORKERS_PER_BLOCK>>>(FILTER_3_OUTPUT, TREECHUNK_FILTER_INPUT);
					// transferResults<<<filter3_numberOfResultsPerRun/ACTUAL_WORKERS_PER_BLOCK + 1, ACTUAL_WORKERS_PER_BLOCK>>>(FILTER_3_OUTPUT, TREECHUNK_FILTER_INPUT);
				#else
					for (uint64_t i = 0; i < filter3_numberOfResultsPerRun; ++i) {
						pthread_create(&threads[i], NULL, transferResults, &data[i]);
					}
				#endif
				TRY_CUDA(hipGetLastError());
				TRY_CUDA(hipDeviceSynchronize());
			}

			treechunkFilter_numberOfResultsPerRun = 0;
			#if CUDA_IS_PRESENT
				// treechunkFilter<<<constexprCeil(static_cast<double>(constexprMin(filter3_numberOfResultsPerRun * (ABSOLUTE_POPULATION_CHUNKS_DATA.getCurrentMaxCalls() + 1) * (UINT64_C(1) << ABSOLUTE_POPULATION_CHUNKS_DATA.getCurrentMaxTreeCount()))/static_cast<double>(ACTUAL_WORKERS_PER_BLOCK)), static_cast<uint64_t>(INT32_MAX)), ACTUAL_WORKERS_PER_BLOCK>>>();
				treechunkFilter<<<constexprMin(filter3_numberOfResultsPerRun * (ABSOLUTE_POPULATION_CHUNKS_DATA.getCurrentMaxCalls() + 1) * (UINT64_C(1) << ABSOLUTE_POPULATION_CHUNKS_DATA.getCurrentMaxTreeCount())/ACTUAL_WORKERS_PER_BLOCK + 1, static_cast<uint64_t>(INT32_MAX)), ACTUAL_WORKERS_PER_BLOCK>>>();
			#else
				for (uint64_t i = 0; i < filter3_numberOfResultsPerRun; ++i) pthread_create(&threads[i], NULL, treechunkFilter, &data[i]); // How to handle larger number of i's?
			#endif
			TRY_CUDA(hipGetLastError());
			TRY_CUDA(hipDeviceSynchronize());

			if (treechunkFilter_numberOfResultsPerRun > ACTUAL_MAX_NUMBER_OF_RESULTS_PER_RUN) {
				if (!SILENT_MODE) std::fprintf(stderr, "WARNING: All-Attributes filter caught more results (%" PRIu64 ") than maximum number allowed (%" PRIu64 "); ignoring last %" PRIu64 " results.\n", treechunkFilter_numberOfResultsPerRun, ACTUAL_MAX_NUMBER_OF_RESULTS_PER_RUN, treechunkFilter_numberOfResultsPerRun - ACTUAL_MAX_NUMBER_OF_RESULTS_PER_RUN);
				treechunkFilter_numberOfResultsPerRun = ACTUAL_MAX_NUMBER_OF_RESULTS_PER_RUN;
			}
			if (!treechunkFilter_numberOfResultsPerRun) continue;

			#if (!DEBUG)
			if (static_cast<void *>(POPULATION_REVERSAL_INPUT) != static_cast<void *>(TREECHUNK_FILTER_OUTPUT)) {
				#if CUDA_IS_PRESENT
					// transferResults<<<constexprCeil(static_cast<double>(treechunkFilter_numberOfResultsPerRun)/static_cast<double>(ACTUAL_WORKERS_PER_BLOCK)), ACTUAL_WORKERS_PER_BLOCK>>>(TREECHUNK_FILTER_OUTPUT, POPULATION_REVERSAL_INPUT);
					// transferResults<<<treechunkFilter_numberOfResultsPerRun/ACTUAL_WORKERS_PER_BLOCK + 1, ACTUAL_WORKERS_PER_BLOCK>>>(TREECHUNK_FILTER_OUTPUT, POPULATION_REVERSAL_INPUT);
				#else
					for (uint64_t i = 0; i < treechunkFilter_numberOfResultsPerRun; ++i) {
						pthread_create(&threads[i], NULL, transferResults, &data[i]);
					}
				#endif
				TRY_CUDA(hipGetLastError());
				TRY_CUDA(hipDeviceSynchronize());
			}
			
			// 1.6.4:         ?
			// 1.8.9:         ?
			// 1.12.2:        ./main input.txt output.txt 1 1.12 treechunkXChunkCoordinate treechunkXChunkCoordinate treechunkZChunkCoordinate treechunkZChunkCoordinate 0 10000
			// 1.14.4:        ./main input.txt output.txt 1 1.13 60001 treechunkXChunkCoordinate treechunkXChunkCoordinate treechunkZChunkCoordinate treechunkZChunkCoordinate 0 0
			// 1.16.1/1.16.4: ./main input.txt output.txt 1 1.13 80001 treechunkXChunkCoordinate treechunkXChunkCoordinate treechunkZChunkCoordinate treechunkZChunkCoordinate 0 0
			totalStructureSeedsPerRun = 0;
			#if CUDA_IS_PRESENT
				// reversePopulationSeeds<<<constexprCeil(static_cast<double>(treechunkFilter_numberOfResultsPerRun)/static_cast<double>(ACTUAL_WORKERS_PER_BLOCK)), ACTUAL_WORKERS_PER_BLOCK>>>();
				reversePopulationSeeds<<<treechunkFilter_numberOfResultsPerRun/ACTUAL_WORKERS_PER_BLOCK + 1, ACTUAL_WORKERS_PER_BLOCK>>>();
			#else
				for (uint64_t i = 0; i < treechunkFilter_numberOfResultsPerRun; ++i) pthread_create(&threads[i], NULL, reversePopulationSeeds, &data[i]);
			#endif
			TRY_CUDA(hipGetLastError());
			TRY_CUDA(hipDeviceSynchronize());

			if (totalStructureSeedsPerRun > ACTUAL_MAX_NUMBER_OF_RESULTS_PER_RUN) {
				if (!SILENT_MODE) std::fprintf(stderr, "WARNING: Population seed reverser caught more results (%" PRIu64 ") than maximum number allowed (%" PRIu64 "); ignoring last %" PRIu64 " results.\n", totalStructureSeedsPerRun, ACTUAL_MAX_NUMBER_OF_RESULTS_PER_RUN, totalStructureSeedsPerRun - ACTUAL_MAX_NUMBER_OF_RESULTS_PER_RUN);
				totalStructureSeedsPerRun = ACTUAL_MAX_NUMBER_OF_RESULTS_PER_RUN;
			}
			if (!totalStructureSeedsPerRun) continue;

			for (currentPopulationChunkDataIndex = 1; currentPopulationChunkDataIndex < ABSOLUTE_POPULATION_CHUNKS_DATA.numberOfTreeChunks; ++currentPopulationChunkDataIndex) {
				if (static_cast<void *>(FILTER_5_INPUT) != (currentPopulationChunkDataIndex == 1 ? static_cast<void *>(POPULATION_REVERSAL_OUTPUT) : static_cast<void *>(FILTER_8_OUTPUT))) {
					// #if CUDA_IS_PRESENT
					// 	// transferResults<<<constexprCeil(static_cast<double>(totalStructureSeedsPerRun)/static_cast<double>(ACTUAL_WORKERS_PER_BLOCK)), ACTUAL_WORKERS_PER_BLOCK>>>(currentPopulationChunkDataIndex == 1 ? POPULATION_REVERSAL_OUTPUT : FILTER_8_OUTPUT, FILTER_5_INPUT);
					// 	transferResults<<<totalStructureSeedsPerRun/ACTUAL_WORKERS_PER_BLOCK + 1, ACTUAL_WORKERS_PER_BLOCK>>>(currentPopulationChunkDataIndex == 1 ? POPULATION_REVERSAL_OUTPUT : FILTER_8_OUTPUT, FILTER_5_INPUT);
						
					// #else
					// 	for (uint64_t i = 0; i < totalStructureSeedsPerRun; ++i) {
					// 		pthread_create(&threads[i], NULL, transferResults, &data[i]);
					// 	}
					// #endif
					// TRY_CUDA(hipGetLastError());
					// TRY_CUDA(hipDeviceSynchronize());
					TRY_CUDA(hipMemcpy(FILTER_5_INPUT, currentPopulationChunkDataIndex == 1 ? POPULATION_REVERSAL_OUTPUT : FILTER_8_OUTPUT, totalStructureSeedsPerRun*(currentPopulationChunkDataIndex == 1 ? sizeof(*POPULATION_REVERSAL_OUTPUT) : sizeof(*FILTER_8_OUTPUT)), hipMemcpyKind::hipMemcpyDefault));
				}

				filter5_numberOfResultsPerRun = 0;
				#if CUDA_IS_PRESENT
					// filter5<<<constexprCeil(static_cast<double>(totalStructureSeedsPerRun)/static_cast<double>(ACTUAL_WORKERS_PER_BLOCK)), ACTUAL_WORKERS_PER_BLOCK>>>();
					filter5<<<totalStructureSeedsPerRun/ACTUAL_WORKERS_PER_BLOCK + 1, ACTUAL_WORKERS_PER_BLOCK>>>();
				#else
					for (uint64_t i = 0; i < totalStructureSeedsPerRun; ++i) pthread_create(&threads[i], NULL, filter5, &data[i]);
				#endif
				TRY_CUDA(hipGetLastError());
				TRY_CUDA(hipDeviceSynchronize());

				if (filter5_numberOfResultsPerRun > ACTUAL_MAX_NUMBER_OF_RESULTS_PER_RUN) {
					if (!SILENT_MODE) std::fprintf(stderr, "WARNING: Structure-to-treechunk Filter caught more results (%" PRIu64 ") than maximum number allowed (%" PRIu64 "); ignoring last %" PRIu64 " results.\n", filter5_numberOfResultsPerRun, ACTUAL_MAX_NUMBER_OF_RESULTS_PER_RUN, filter5_numberOfResultsPerRun - ACTUAL_MAX_NUMBER_OF_RESULTS_PER_RUN);
					filter5_numberOfResultsPerRun = ACTUAL_MAX_NUMBER_OF_RESULTS_PER_RUN;
				}
				if (!filter5_numberOfResultsPerRun) continue;

				
				if (static_cast<void *>(FILTER_6_INPUT) != static_cast<void *>(FILTER_5_OUTPUT)) {
					#if CUDA_IS_PRESENT
						// transferResults<<<constexprCeil(static_cast<double>(filter5_numberOfResultsPerRun)/static_cast<double>(ACTUAL_WORKERS_PER_BLOCK)), ACTUAL_WORKERS_PER_BLOCK>>>(FILTER_5_OUTPUT, FILTER_6_INPUT);
						// transferResults<<<filter5_numberOfResultsPerRun/ACTUAL_WORKERS_PER_BLOCK + 1, ACTUAL_WORKERS_PER_BLOCK>>>(FILTER_5_OUTPUT, FILTER_6_INPUT);
					#else
						for (uint64_t i = 0; i < filter5_numberOfResultsPerRun; ++i) {
							pthread_create(&threads[i], NULL, transferResults, &data[i]);
						}
					#endif
					TRY_CUDA(hipGetLastError());
					TRY_CUDA(hipDeviceSynchronize());
				}

				filter6_numberOfResultsPerRun = 0;
				#if CUDA_IS_PRESENT
					// filter6<<<constexprCeil(static_cast<double>(filter5_numberOfResultsPerRun)/static_cast<double>(ACTUAL_WORKERS_PER_BLOCK)), ACTUAL_WORKERS_PER_BLOCK>>>();
					filter6<<<filter5_numberOfResultsPerRun/ACTUAL_WORKERS_PER_BLOCK + 1, ACTUAL_WORKERS_PER_BLOCK>>>();
				#else
					for (uint64_t i = 0; i < filter5_numberOfResultsPerRun; ++i) pthread_create(&threads[i], NULL, filter6, &data[i]);
				#endif
				TRY_CUDA(hipGetLastError());
				TRY_CUDA(hipDeviceSynchronize());

				if (filter6_numberOfResultsPerRun > ACTUAL_MAX_NUMBER_OF_RESULTS_PER_RUN) {
					if (!SILENT_MODE) std::fprintf(stderr, "WARNING: Structure-to-treechunk Filter caught more results (%" PRIu64 ") than maximum number allowed (%" PRIu64 "); ignoring last %" PRIu64 " results.\n", filter6_numberOfResultsPerRun, ACTUAL_MAX_NUMBER_OF_RESULTS_PER_RUN, filter6_numberOfResultsPerRun - ACTUAL_MAX_NUMBER_OF_RESULTS_PER_RUN);
					filter6_numberOfResultsPerRun = ACTUAL_MAX_NUMBER_OF_RESULTS_PER_RUN;
				}
				if (!filter6_numberOfResultsPerRun) continue;


				if (static_cast<void *>(FILTER_7_INPUT) != static_cast<void *>(FILTER_6_OUTPUT)) {
					#if CUDA_IS_PRESENT
						// transferResults<<<constexprCeil(static_cast<double>(filter6_numberOfResultsPerRun)/static_cast<double>(ACTUAL_WORKERS_PER_BLOCK)), ACTUAL_WORKERS_PER_BLOCK>>>(FILTER_6_OUTPUT, FILTER_7_INPUT);
						// transferResults<<<filter6_numberOfResultsPerRun/ACTUAL_WORKERS_PER_BLOCK + 1, ACTUAL_WORKERS_PER_BLOCK>>>(FILTER_6_OUTPUT, FILTER_7_INPUT);
					#else
						for (uint64_t i = 0; i < filter6_numberOfResultsPerRun; ++i) {
							pthread_create(&threads[i], NULL, transferResults, &data[i]);
						}
					#endif
					TRY_CUDA(hipGetLastError());
					TRY_CUDA(hipDeviceSynchronize());
				}

				filter7_numberOfResultsPerRun = 0;
				#if CUDA_IS_PRESENT
					// filter7<<<constexprCeil(static_cast<double>(filter6_numberOfResultsPerRun)/static_cast<double>(ACTUAL_WORKERS_PER_BLOCK)), ACTUAL_WORKERS_PER_BLOCK>>>();
					filter7<<<filter6_numberOfResultsPerRun/ACTUAL_WORKERS_PER_BLOCK + 1, ACTUAL_WORKERS_PER_BLOCK>>>();
				#else
					for (uint64_t i = 0; i < filter6_numberOfResultsPerRun; ++i) pthread_create(&threads[i], NULL, filter7, &data[i]);
				#endif
				TRY_CUDA(hipGetLastError());
				TRY_CUDA(hipDeviceSynchronize());

				if (filter7_numberOfResultsPerRun > ACTUAL_MAX_NUMBER_OF_RESULTS_PER_RUN) {
					if (!SILENT_MODE) std::fprintf(stderr, "WARNING: Structure-to-treechunk Filter caught more results (%" PRIu64 ") than maximum number allowed (%" PRIu64 "); ignoring last %" PRIu64 " results.\n", filter7_numberOfResultsPerRun, ACTUAL_MAX_NUMBER_OF_RESULTS_PER_RUN, filter7_numberOfResultsPerRun - ACTUAL_MAX_NUMBER_OF_RESULTS_PER_RUN);
					filter7_numberOfResultsPerRun = ACTUAL_MAX_NUMBER_OF_RESULTS_PER_RUN;
				}
				if (!filter7_numberOfResultsPerRun) continue;


				if (static_cast<void *>(FILTER_8_STRUCTURESEED_INPUT) != (currentPopulationChunkDataIndex == 1 ? static_cast<void *>(POPULATION_REVERSAL_OUTPUT) : static_cast<void *>(FILTER_8_OUTPUT))) {
					// #if CUDA_IS_PRESENT
					// 	// transferResults<<<constexprCeil(static_cast<double>(totalStructureSeedsPerRun)/static_cast<double>(ACTUAL_WORKERS_PER_BLOCK)), ACTUAL_WORKERS_PER_BLOCK>>>(currentPopulationChunkDataIndex == 1 ? POPULATION_REVERSAL_OUTPUT : FILTER_8_OUTPUT, FILTER_8_STRUCTURESEED_INPUT);
					// 	transferResults<<<totalStructureSeedsPerRun/ACTUAL_WORKERS_PER_BLOCK + 1, ACTUAL_WORKERS_PER_BLOCK>>>(currentPopulationChunkDataIndex == 1 ? POPULATION_REVERSAL_OUTPUT : FILTER_8_OUTPUT, FILTER_8_STRUCTURESEED_INPUT);
					// #else
					// 	for (uint64_t i = 0; i < totalStructureSeedsPerRun; ++i) {
					// 		pthread_create(&threads[i], NULL, transferResults, &data[i]);
					// 	}
					// #endif
					// TRY_CUDA(hipGetLastError());
					// TRY_CUDA(hipDeviceSynchronize());
					TRY_CUDA(hipMemcpy(FILTER_8_STRUCTURESEED_INPUT, currentPopulationChunkDataIndex == 1 ? POPULATION_REVERSAL_OUTPUT : FILTER_8_OUTPUT, totalStructureSeedsPerRun*(currentPopulationChunkDataIndex == 1 ? sizeof(*POPULATION_REVERSAL_OUTPUT) : sizeof(*FILTER_8_OUTPUT)), hipMemcpyKind::hipMemcpyDefault));
				}
				if (static_cast<void *>(FILTER_8_TREECHUNK_INPUT) != static_cast<void *>(FILTER_7_OUTPUT)) {
					#if CUDA_IS_PRESENT
						// transferResults<<<constexprCeil(static_cast<double>(filter7_numberOfResultsPerRun)/static_cast<double>(ACTUAL_WORKERS_PER_BLOCK)), ACTUAL_WORKERS_PER_BLOCK>>>(FILTER_7_OUTPUT, FILTER_8_TREECHUNK_INPUT);
						// transferResults<<<filter7_numberOfResultsPerRun/ACTUAL_WORKERS_PER_BLOCK + 1, ACTUAL_WORKERS_PER_BLOCK>>>(FILTER_7_OUTPUT, FILTER_8_TREECHUNK_INPUT);
					#else
						for (uint64_t i = 0; i < filter7_numberOfResultsPerRun; ++i) {
							pthread_create(&threads[i], NULL, transferResults, &data[i]);
						}
					#endif
					TRY_CUDA(hipGetLastError());
					TRY_CUDA(hipDeviceSynchronize());
				}

				totalStructureSeedsPerRun = 0;
				#if CUDA_IS_PRESENT
					// filter8<<<constexprCeil(static_cast<double>(filter7_numberOfResultsPerRun)/static_cast<double>(ACTUAL_WORKERS_PER_BLOCK)), ACTUAL_WORKERS_PER_BLOCK>>>();
					filter8<<<filter7_numberOfResultsPerRun/ACTUAL_WORKERS_PER_BLOCK + 1, ACTUAL_WORKERS_PER_BLOCK>>>();
				#else
					for (uint64_t i = 0; i < filter7_numberOfResultsPerRun; ++i) pthread_create(&threads[i], NULL, filter8, &data[i]);
				#endif
				TRY_CUDA(hipGetLastError());
				TRY_CUDA(hipDeviceSynchronize());

				if (totalStructureSeedsPerRun > ACTUAL_MAX_NUMBER_OF_RESULTS_PER_RUN) {
					if (!SILENT_MODE) std::fprintf(stderr, "WARNING: Filter 8 caught more results (%" PRIu64 ") than maximum number allowed (%" PRIu64 "); ignoring last %" PRIu64 " results.\n", totalStructureSeedsPerRun, ACTUAL_MAX_NUMBER_OF_RESULTS_PER_RUN, totalStructureSeedsPerRun - ACTUAL_MAX_NUMBER_OF_RESULTS_PER_RUN);
					totalStructureSeedsPerRun = ACTUAL_MAX_NUMBER_OF_RESULTS_PER_RUN;
				}
				if (!totalStructureSeedsPerRun) continue;
			}

			// Pass all structure seeds through a set to remove duplicates (and also order them)
			std::set<uint64_t> uniqueStructureSeeds;
			for (uint64_t i = 0; i < totalStructureSeedsPerRun; ++i) uniqueStructureSeeds.insert(ABSOLUTE_POPULATION_CHUNKS_DATA.numberOfTreeChunks == 1 ? POPULATION_REVERSAL_OUTPUT[i] : FILTER_8_OUTPUT[i]);
			totalStructureSeedsPerRun = static_cast<uint64_t>(uniqueStructureSeeds.size());
			uint64_t count = 0;
			for (auto i = uniqueStructureSeeds.cbegin(); i != uniqueStructureSeeds.cend(); ++i) (ABSOLUTE_POPULATION_CHUNKS_DATA.numberOfTreeChunks == 1 ? POPULATION_REVERSAL_OUTPUT[count++] : FILTER_8_OUTPUT[count++]) = *i;
			uniqueStructureSeeds.clear();
			#endif

			// if (static_cast<void *>(WORLDSEED_FILTER_INPUT) != (ABSOLUTE_POPULATION_CHUNKS_DATA.numberOfTreeChunks == 1 ? static_cast<void *>(POPULATION_REVERSAL_OUTPUT) : static_cast<void *>(FILTER_8_OUTPUT)))
			// #if CUDA_IS_PRESENT
			// 	// transferResults<<<constexprCeil(static_cast<double>(totalStructureSeedsPerRun)/static_cast<double>(ACTUAL_WORKERS_PER_BLOCK)), ACTUAL_WORKERS_PER_BLOCK>>>(ABSOLUTE_POPULATION_CHUNKS_DATA.numberOfTreeChunks == 1 ? POPULATION_REVERSAL_OUTPUT : FILTER_8_OUTPUT, WORLDSEED_FILTER_INPUT);
			// 	transferResults<<<totalStructureSeedsPerRun/ACTUAL_WORKERS_PER_BLOCK + 1, ACTUAL_WORKERS_PER_BLOCK>>>(ABSOLUTE_POPULATION_CHUNKS_DATA.numberOfTreeChunks == 1 ? POPULATION_REVERSAL_OUTPUT : FILTER_8_OUTPUT, WORLDSEED_FILTER_INPUT);
			// #else
			// 	for (uint64_t i = 0; i < totalStructureSeedsPerRun; ++i) {
			// 		pthread_create(&threads[i], NULL, transferResults, &data[i]);
			// 	}
			// #endif
			// TRY_CUDA(hipGetLastError());
			// TRY_CUDA(hipDeviceSynchronize());

			// for (largeBiomesFlag = 0; largeBiomesFlag <= 1; ++largeBiomesFlag) {
			// 	totalWorldseedsPerRun = 0;
			// 	#if CUDA_IS_PRESENT
			// 		// biomeFilter<<<constexprMin(constexprCeil(static_cast<double>(totalStructureSeedsPerRun*65536)/static_cast<double>(ACTUAL_WORKERS_PER_BLOCK)), static_cast<uint64_t>(INT32_MAX)), ACTUAL_WORKERS_PER_BLOCK>>>();
			// 		biomeFilter<<<constexprMin(totalStructureSeedsPerRun*65536/ACTUAL_WORKERS_PER_BLOCK + 1, static_cast<uint64_t>(INT32_MAX)), ACTUAL_WORKERS_PER_BLOCK>>>();
			// 	#else
			// 		for (uint64_t i = 0; i < totalStructureSeedsPerRun; ++i) pthread_create(&threads[i], NULL, biomeFilter, &data[i]);
			// 	#endif
			// 	TRY_CUDA(hipGetLastError());
			// 	TRY_CUDA(hipDeviceSynchronize());

			// 	if (totalWorldseedsPerRun > ACTUAL_MAX_NUMBER_OF_RESULTS_PER_RUN) {
			// 		if (!SILENT_MODE) std::fprintf(stderr, "WARNING: Biome filter caught more results (%" PRIu64 ") than maximum number allowed (%" PRIu64 "); ignoring last %" PRIu64 " results.\n", totalWorldseedsPerRun, ACTUAL_MAX_NUMBER_OF_RESULTS_PER_RUN, totalWorldseedsPerRun - ACTUAL_MAX_NUMBER_OF_RESULTS_PER_RUN);
			// 		totalWorldseedsPerRun = ACTUAL_MAX_NUMBER_OF_RESULTS_PER_RUN;
			// 	}
			// 	for (uint64_t i = 0; i < totalWorldseedsPerRun; ++i) {
				// const char *notes = getNotesAboutSeed(WORLDSEED_FILTER_OUTPUT[i], largeBiomesFlag);
				// if (OUTPUT_FILEPATH != NULL && OUTPUT_FILEPATH != "") std::fprintf(outputFile, "%" PRId64"\t%s\n", static_cast<int64_t>(WORLDSEED_FILTER_OUTPUT[i]), notes);
				// if (!SILENT_MODE) std::printf("%" PRId64 "\t%s\n", static_cast<int64_t>(WORLDSEED_FILTER_OUTPUT[i]), notes);
			// }

			#if DEBUG
			for (uint64_t i = 0; i < treechunkFilter_numberOfResultsPerRun; ++i) {
				if (OUTPUT_FILEPATH != NULL && OUTPUT_FILEPATH != "") std::fprintf(outputFile, "%" PRId64"\n", static_cast<int64_t>(TREECHUNK_FILTER_OUTPUT[i]));
				if (!SILENT_MODE) std::printf("%" PRId64 "\n", static_cast<int64_t>(TREECHUNK_FILTER_OUTPUT[i]));
			}
			#else
			for (uint64_t i = 0; i < totalStructureSeedsPerRun; ++i) {
				if (OUTPUT_FILEPATH != NULL && OUTPUT_FILEPATH != "") std::fprintf(outputFile, "%" PRId64"\n", static_cast<int64_t>(POPULATION_REVERSAL_OUTPUT[i]));
				if (!SILENT_MODE) std::printf("%" PRId64 "\n", static_cast<int64_t>(POPULATION_REVERSAL_OUTPUT[i]));
			}
			#endif
		}
	}

	if (OUTPUT_FILEPATH != NULL && OUTPUT_FILEPATH != "") std::fclose(outputFile);

	#if (!CUDA_IS_PRESENT)
		free(threads);
	#endif
	return 0;
}